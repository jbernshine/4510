#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "vec_mult.h"
#include "utils.h"
#include "kernels.h"

void print_results(int n, int block_size, int blocks)
{
    printf("-------\n");
    printf("Results\n");
    printf("-------\n");
    
    // Print the resulting C vector and the timing stats
    printf("Data size: 2^%d = %d\n", (int) log2f(n), n);
    printf("blocks_size: %d\n", block_size);
    printf("blocks: %d\n", blocks);
    printf("\n");

}

int main(int argc, char *argv[])
{
	// All CUDA API calls return a status, which we must check
    hipError_t statusA; // records status of operations on A vec
    hipError_t statusB; // for B vec
    hipError_t statusC; // for C vec

	// Allocate host buffers
    size_t size = N * sizeof(float); //size in bytes
	size_t size_a = N * N * sizeof(float); //size in bytes
    float *host_a = (float *) malloc(size_a);
    float *host_b = (float *) malloc(size);
    float *host_c = (float *) malloc(size);

	// Fill host buffers with integers and print them
    init_vec(host_a, (N * N));
    init_vec(host_b, N);
    print_vec("A vector:\n", host_a, (N * N));
    print_vec("B vector:\n", host_b, N);

	// Allocate device buffers
    float *dev_a;
    float *dev_b;
    float *dev_c;
    statusA = hipMalloc(&dev_a, size_a);
    check_error(statusA, "Error allocating dev buffer A.");
    statusB = hipMalloc(&dev_b, size);
    check_error(statusB, "Error allocating dev buffer B.");
    statusC = hipMalloc(&dev_c, size);
    check_error(statusC, "Error allocating dev buffer C.");

	// Transfer the input vectors from host to device
	statusA = hipMemcpy(dev_a, host_a, size_a, hipMemcpyHostToDevice);
    statusB = hipMemcpy(dev_b, host_b, size, hipMemcpyHostToDevice);

	check_error(statusA, "Error on CPU->GPU hipMemcpy for A.");
    check_error(statusB, "Error on CPU->GPU hipMemcpy for B.");
	
	// Determine number of blocks to launch
    // We'll use the max possible number of threads per block
    //int block_size = get_max_block_threads();
    // We'll allocate n threads (one to add each column in the vectors).
    // If n is not evenly divisible by block_size, we'll need to launch one more
    // block to handle the extras (note: not all its threads will be active)
    //int blocks = N / block_size + (N % block_size > 0 ? 1 : 0);
	
	// 1 block with 256 threads
	multiply<<<1, 256>>>(dev_a, dev_b);

	// n blocks with n / 2 threads
	reduce<<<N, (N / 2)>>>(dev_a, dev_c, N);

	// Transfer the resulting C vector back to the host
	statusC = hipMemcpy(host_c, dev_c, size, hipMemcpyDeviceToHost);
	check_error(statusC, "Error on GPU->CPU hipMemcpy for C.");

	// Display the result
	print_vec("C vector:\n", host_c, N);
	//print_results(N, block_size, blocks);

	// Clean up memory on host
    free(host_a);
    free(host_b);
    free(host_c);

	// Clean up memory on device
    statusA = hipFree(dev_a);
    statusB = hipFree(dev_b);
    statusC = hipFree(dev_c);
    check_error(statusA, "Error calling hipFree on dev_a buffer" );
    check_error(statusB, "Error calling hipFree on dev_b buffer" );
    check_error(statusC, "Error calling hipFree on dev_c buffer" );

	return EXIT_SUCCESS;
}
